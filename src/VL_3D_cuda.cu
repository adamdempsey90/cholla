
#include <hip/hip_runtime.h>
/*! \file VL_3D_cuda.cu
 *  \brief Definitions of the cuda 3D VL algorithm functions. */

#ifdef CUDA
#ifdef VL

#include<stdio.h>
#include<stdlib.h>
#include<math.h>
#include<cuda.h>
#include"global.h"
#include"global_cuda.h"
#include"hydro_cuda.h"
#include"VL_3D_cuda.h"
#include"pcm_cuda.h"
#include"plmp_vl_cuda.h"
#include"plmc_vl_cuda.h"
#include"ppmp_vl_cuda.h"
#include"ppmc_vl_cuda.h"
#include"exact_cuda.h"
#include"roe_cuda.h"
#include"h_correction_3D_cuda.h"
#include"cooling_cuda.h"
#include"subgrid_routines_3D.h"


__global__ void Update_Conserved_Variables_3D_half(Real *dev_conserved, Real *dev_conserved_half, Real *dev_F_x, Real *dev_F_y,  Real *dev_F_z,
                                              int nx, int ny, int nz, int n_ghost, Real dx, Real dy, Real dz, Real dt, Real gamma);



Real VL_Algorithm_3D_CUDA(Real *host_conserved, int nx, int ny, int nz, int n_ghost, Real dx, Real dy, Real dz, Real dt)
{

  //Here, *host_conserved contains the entire
  //set of conserved variables on the grid
  //concatenated into a 1-d array

  int n_fields = 5;
  #ifdef DE
  n_fields++;
  #endif

  // dimensions of subgrid blocks
  int nx_s; //number of cells in the subgrid block along x direction
  int ny_s; //number of cells in the subgrid block along y direction
  int nz_s; //number of cells in the subgrid block along z direction

  // total number of blocks needed
  int block_tot;    //total number of subgrid blocks (unsplit == 1)
  int block1_tot;   //total number of subgrid blocks in x direction
  int block2_tot;   //total number of subgrid blocks in y direction
  int block3_tot;   //total number of subgrid blocks in z direction 
  int remainder1;   //modulus of number of cells after block subdivision in x direction
  int remainder2;   //modulus of number of cells after block subdivision in y direction 
  int remainder3;   //modulus of number of cells after block subdivision in z direction

  // counter for which block we're on
  int block = 0;

  // calculate the dimensions for each subgrid block
  sub_dimensions_3D(nx, ny, nz, n_ghost, &nx_s, &ny_s, &nz_s, &block1_tot, &block2_tot, &block3_tot, &remainder1, &remainder2, &remainder3, n_fields);
  block_tot = block1_tot*block2_tot*block3_tot;

  // number of cells in one subgrid block
  int BLOCK_VOL = nx_s*ny_s*nz_s;


  // define the dimensions for the 1D grid
  int  ngrid = (BLOCK_VOL + TPB - 1) / TPB;

  //number of blocks per 1-d grid  
  dim3 dim1dGrid(ngrid, 1, 1);

  //number of threads per 1-d block   
  dim3 dim1dBlock(TPB, 1, 1);


  // allocate buffer arrays to copy conserved variable slices into
  Real **buffer;
  allocate_buffers_3D(block1_tot, block2_tot, block3_tot, BLOCK_VOL, buffer, n_fields);
  // and set up pointers for the location to copy from and to
  Real *tmp1;
  Real *tmp2;


  // allocate an array on the CPU to hold max_dti returned from each thread block
  Real max_dti = 0;
  Real *host_dti_array;
  host_dti_array = (Real *) malloc(ngrid*sizeof(Real));

  // allocate GPU arrays
  // conserved variables
  Real *dev_conserved, *dev_conserved_half;
  // input states and associated interface fluxes (Q* and F* from Stone, 2008)
  Real *Q_Lx, *Q_Rx, *Q_Ly, *Q_Ry, *Q_Lz, *Q_Rz, *F_x, *F_y, *F_z;
  // arrays to hold the eta values for the H correction
  Real *eta_x, *eta_y, *eta_z, *etah_x, *etah_y, *etah_z;
  // array of inverse timesteps for dt calculation
  Real *dev_dti_array;

  // allocate memory on the GPU
  CudaSafeCall( cudaMalloc((void**)&dev_conserved, n_fields*BLOCK_VOL*sizeof(Real)) );
  CudaSafeCall( cudaMalloc((void**)&dev_conserved_half, n_fields*BLOCK_VOL*sizeof(Real)) );
  CudaSafeCall( cudaMalloc((void**)&Q_Lx,  n_fields*BLOCK_VOL*sizeof(Real)) );
  CudaSafeCall( cudaMalloc((void**)&Q_Rx,  n_fields*BLOCK_VOL*sizeof(Real)) );
  CudaSafeCall( cudaMalloc((void**)&Q_Ly,  n_fields*BLOCK_VOL*sizeof(Real)) );
  CudaSafeCall( cudaMalloc((void**)&Q_Ry,  n_fields*BLOCK_VOL*sizeof(Real)) );
  CudaSafeCall( cudaMalloc((void**)&Q_Lz,  n_fields*BLOCK_VOL*sizeof(Real)) );
  CudaSafeCall( cudaMalloc((void**)&Q_Rz,  n_fields*BLOCK_VOL*sizeof(Real)) );
  CudaSafeCall( cudaMalloc((void**)&F_x,   n_fields*BLOCK_VOL*sizeof(Real)) );
  CudaSafeCall( cudaMalloc((void**)&F_y,   n_fields*BLOCK_VOL*sizeof(Real)) );
  CudaSafeCall( cudaMalloc((void**)&F_z,   n_fields*BLOCK_VOL*sizeof(Real)) );
  CudaSafeCall( cudaMalloc((void**)&eta_x,  BLOCK_VOL*sizeof(Real)) );
  CudaSafeCall( cudaMalloc((void**)&eta_y,  BLOCK_VOL*sizeof(Real)) );
  CudaSafeCall( cudaMalloc((void**)&eta_z,  BLOCK_VOL*sizeof(Real)) );
  CudaSafeCall( cudaMalloc((void**)&etah_x, BLOCK_VOL*sizeof(Real)) );
  CudaSafeCall( cudaMalloc((void**)&etah_y, BLOCK_VOL*sizeof(Real)) );
  CudaSafeCall( cudaMalloc((void**)&etah_z, BLOCK_VOL*sizeof(Real)) );
  CudaSafeCall( cudaMalloc((void**)&dev_dti_array, ngrid*sizeof(Real)) );


  // transfer first conserved variable slice into the first buffer
  host_copy_init_3D(nx, ny, nz, nx_s, ny_s, nz_s, n_ghost, block, block1_tot, block2_tot, remainder1, remainder2, BLOCK_VOL, host_conserved, buffer, &tmp1, &tmp2, n_fields);


  // START LOOP OVER SUBGRID BLOCKS HERE
  while (block < block_tot) {

  // zero the GPU arrays
  cudaMemset(dev_conserved, 0, n_fields*BLOCK_VOL*sizeof(Real));
  cudaMemset(dev_conserved_half, 0, n_fields*BLOCK_VOL*sizeof(Real));
  cudaMemset(Q_Lx,  0, n_fields*BLOCK_VOL*sizeof(Real));
  cudaMemset(Q_Rx,  0, n_fields*BLOCK_VOL*sizeof(Real));
  cudaMemset(Q_Ly,  0, n_fields*BLOCK_VOL*sizeof(Real));
  cudaMemset(Q_Ry,  0, n_fields*BLOCK_VOL*sizeof(Real));
  cudaMemset(Q_Lz,  0, n_fields*BLOCK_VOL*sizeof(Real));
  cudaMemset(Q_Rz,  0, n_fields*BLOCK_VOL*sizeof(Real));
  cudaMemset(F_x,   0, n_fields*BLOCK_VOL*sizeof(Real));
  cudaMemset(F_y,   0, n_fields*BLOCK_VOL*sizeof(Real));
  cudaMemset(F_z,   0, n_fields*BLOCK_VOL*sizeof(Real));
  cudaMemset(eta_x,  0, BLOCK_VOL*sizeof(Real));
  cudaMemset(eta_y,  0, BLOCK_VOL*sizeof(Real));
  cudaMemset(eta_z,  0, BLOCK_VOL*sizeof(Real));
  cudaMemset(etah_x, 0, BLOCK_VOL*sizeof(Real));
  cudaMemset(etah_y, 0, BLOCK_VOL*sizeof(Real));
  cudaMemset(etah_z, 0, BLOCK_VOL*sizeof(Real));
  cudaMemset(dev_dti_array, 0, ngrid*sizeof(Real));  
  CudaCheckError();


  // copy the conserved variables onto the GPU
  CudaSafeCall( cudaMemcpy(dev_conserved, tmp1, n_fields*BLOCK_VOL*sizeof(Real), cudaMemcpyHostToDevice) );
  

  // Step 1: Use PCM reconstruction to put primitive variables into interface arrays
  PCM_Reconstruction_3D<<<dim1dGrid,dim1dBlock>>>(dev_conserved, Q_Lx, Q_Rx, Q_Ly, Q_Ry, Q_Lz, Q_Rz, nx_s, ny_s, nz_s, n_ghost, gama);
  CudaCheckError();


  // Step 2: Calculate first-order upwind fluxes 
  #ifdef EXACT
  Calculate_Exact_Fluxes<<<dim1dGrid,dim1dBlock>>>(Q_Lx, Q_Rx, F_x, nx_s, ny_s, nz_s, n_ghost, gama, 0);
  Calculate_Exact_Fluxes<<<dim1dGrid,dim1dBlock>>>(Q_Ly, Q_Ry, F_y, nx_s, ny_s, nz_s, n_ghost, gama, 1);
  Calculate_Exact_Fluxes<<<dim1dGrid,dim1dBlock>>>(Q_Lz, Q_Rz, F_z, nx_s, ny_s, nz_s, n_ghost, gama, 2);
  #endif //EXACT
  #ifdef ROE
  Calculate_Roe_Fluxes<<<dim1dGrid,dim1dBlock>>>(Q_Lx, Q_Rx, F_x, nx_s, ny_s, nz_s, n_ghost, gama, etah_x, 0);
  Calculate_Roe_Fluxes<<<dim1dGrid,dim1dBlock>>>(Q_Ly, Q_Ry, F_y, nx_s, ny_s, nz_s, n_ghost, gama, etah_y, 1);
  Calculate_Roe_Fluxes<<<dim1dGrid,dim1dBlock>>>(Q_Lz, Q_Rz, F_z, nx_s, ny_s, nz_s, n_ghost, gama, etah_z, 2);
  #endif //ROE
  CudaCheckError();


  // Step 3: Update the conserved variables half a timestep 
  Update_Conserved_Variables_3D_half<<<dim1dGrid,dim1dBlock>>>(dev_conserved, dev_conserved_half, F_x, F_y, F_z, nx_s, ny_s, nz_s, n_ghost, dx, dy, dz, 0.5*dt, gama);
  CudaCheckError();


  // Step 4: Construct left and right interface values using updated conserved variables
  #ifdef PCM
  PCM_Reconstruction_3D<<<dim1dGrid,dim1dBlock>>>(dev_conserved_half, Q_Lx, Q_Rx, Q_Ly, Q_Ry, Q_Lz, Q_Rz, nx_s, ny_s, nz_s, n_ghost, gama);
  #endif
  #ifdef PLMP
  PLMP_VL<<<dim1dGrid,dim1dBlock>>>(dev_conserved_half, Q_Lx, Q_Rx, nx_s, ny_s, nz_s, n_ghost, gama, 0);
  PLMP_VL<<<dim1dGrid,dim1dBlock>>>(dev_conserved_half, Q_Ly, Q_Ry, nx_s, ny_s, nz_s, n_ghost, gama, 1);
  PLMP_VL<<<dim1dGrid,dim1dBlock>>>(dev_conserved_half, Q_Lz, Q_Rz, nx_s, ny_s, nz_s, n_ghost, gama, 2);
  #endif //PLMP 
  #ifdef PLMC
  PLMC_VL<<<dim1dGrid,dim1dBlock>>>(dev_conserved_half, Q_Lx, Q_Rx, nx_s, ny_s, nz_s, n_ghost, gama, 0);
  PLMC_VL<<<dim1dGrid,dim1dBlock>>>(dev_conserved_half, Q_Ly, Q_Ry, nx_s, ny_s, nz_s, n_ghost, gama, 1);
  PLMC_VL<<<dim1dGrid,dim1dBlock>>>(dev_conserved_half, Q_Lz, Q_Rz, nx_s, ny_s, nz_s, n_ghost, gama, 2);  
  #endif
  #ifdef PPMP
  PPMP_VL<<<dim1dGrid,dim1dBlock>>>(dev_conserved_half, Q_Lx, Q_Rx, nx_s, ny_s, nz_s, n_ghost, gama, 0);
  PPMP_VL<<<dim1dGrid,dim1dBlock>>>(dev_conserved_half, Q_Ly, Q_Ry, nx_s, ny_s, nz_s, n_ghost, gama, 1);
  PPMP_VL<<<dim1dGrid,dim1dBlock>>>(dev_conserved_half, Q_Lz, Q_Rz, nx_s, ny_s, nz_s, n_ghost, gama, 2);
  #endif //PPMP
  #ifdef PPMC
  PPMC_VL<<<dim1dGrid,dim1dBlock>>>(dev_conserved_half, Q_Lx, Q_Rx, nx_s, ny_s, nz_s, n_ghost, gama, 0);
  PPMC_VL<<<dim1dGrid,dim1dBlock>>>(dev_conserved_half, Q_Ly, Q_Ry, nx_s, ny_s, nz_s, n_ghost, gama, 1);
  PPMC_VL<<<dim1dGrid,dim1dBlock>>>(dev_conserved_half, Q_Lz, Q_Rz, nx_s, ny_s, nz_s, n_ghost, gama, 2);
  #endif //PPMC
  CudaCheckError();
  

  #ifdef H_CORRECTION
  // Step 4.5: Calculate eta values for H correction
  calc_eta_x_3D<<<dim1dGrid,dim1dBlock>>>(Q_Lx, Q_Rx, eta_x, nx_s, ny_s, nz_s, n_ghost, gama);
  calc_eta_y_3D<<<dim1dGrid,dim1dBlock>>>(Q_Ly, Q_Ry, eta_y, nx_s, ny_s, nz_s, n_ghost, gama);
  calc_eta_z_3D<<<dim1dGrid,dim1dBlock>>>(Q_Lz, Q_Rz, eta_z, nx_s, ny_s, nz_s, n_ghost, gama);
  CudaCheckError();
  // and etah values for each interface
  calc_etah_x_3D<<<dim1dGrid,dim1dBlock>>>(eta_x, eta_y, eta_z, etah_x, nx_s, ny_s, nz_s, n_ghost);
  calc_etah_y_3D<<<dim1dGrid,dim1dBlock>>>(eta_x, eta_y, eta_z, etah_y, nx_s, ny_s, nz_s, n_ghost);
  calc_etah_z_3D<<<dim1dGrid,dim1dBlock>>>(eta_x, eta_y, eta_z, etah_z, nx_s, ny_s, nz_s, n_ghost);
  CudaCheckError();
  #endif //H_CORRECTION


  // Step 5: Calculate the fluxes again
  #ifdef EXACT
  Calculate_Exact_Fluxes<<<dim1dGrid,dim1dBlock>>>(Q_Lx, Q_Rx, F_x, nx_s, ny_s, nz_s, n_ghost, gama, 0);
  Calculate_Exact_Fluxes<<<dim1dGrid,dim1dBlock>>>(Q_Ly, Q_Ry, F_y, nx_s, ny_s, nz_s, n_ghost, gama, 1);
  Calculate_Exact_Fluxes<<<dim1dGrid,dim1dBlock>>>(Q_Lz, Q_Rz, F_z, nx_s, ny_s, nz_s, n_ghost, gama, 2);
  #endif //EXACT
  #ifdef ROE
  Calculate_Roe_Fluxes<<<dim1dGrid,dim1dBlock>>>(Q_Lx, Q_Rx, F_x, nx_s, ny_s, nz_s, n_ghost, gama, etah_x, 0);
  Calculate_Roe_Fluxes<<<dim1dGrid,dim1dBlock>>>(Q_Ly, Q_Ry, F_y, nx_s, ny_s, nz_s, n_ghost, gama, etah_y, 1);
  Calculate_Roe_Fluxes<<<dim1dGrid,dim1dBlock>>>(Q_Lz, Q_Rz, F_z, nx_s, ny_s, nz_s, n_ghost, gama, etah_z, 2);
  #endif //ROE
  CudaCheckError();


  // Step 6: Update the conserved variable array
  Update_Conserved_Variables_3D<<<dim1dGrid,dim1dBlock>>>(dev_conserved, F_x, F_y, F_z, nx_s, ny_s, nz_s, n_ghost, dx, dy, dz, dt, gama);
  CudaCheckError();

  #ifdef DE
  Sync_Energies_3D<<<dim1dGrid,dim1dBlock>>>(dev_conserved, nx_s, ny_s, nz_s, n_ghost, gama);
  #endif

  // Apply cooling
  #ifdef COOLING_GPU
  cooling_kernel<<<dim1dGrid,dim1dBlock>>>(dev_conserved, nx_s, ny_s, nz_s, n_ghost, dt, gama);
  #endif

  
  // Step 7: Calculate the next time step
  Calc_dt_3D<<<dim1dGrid,dim1dBlock>>>(dev_conserved, nx_s, ny_s, nz_s, n_ghost, dx, dy, dz, dev_dti_array, gama);
  CudaCheckError();

  // copy the updated conserved variable array back to the CPU
  CudaSafeCall( cudaMemcpy(tmp2, dev_conserved, n_fields*BLOCK_VOL*sizeof(Real), cudaMemcpyDeviceToHost) );

  // copy the next conserved variable blocks into appropriate buffers
  host_copy_next_3D(nx, ny, nz, nx_s, ny_s, nz_s, n_ghost, block, block1_tot, block2_tot, block3_tot, remainder1, remainder2, remainder3, BLOCK_VOL, host_conserved, buffer, &tmp1, n_fields);

  // copy the updated conserved variable array back into the host_conserved array on the CPU
  host_return_values_3D(nx, ny, nz, nx_s, ny_s, nz_s, n_ghost, block, block1_tot, block2_tot, block3_tot, remainder1, remainder2, remainder3, BLOCK_VOL, host_conserved, buffer, n_fields);


  // copy the dti array onto the CPU
  CudaSafeCall( cudaMemcpy(host_dti_array, dev_dti_array, ngrid*sizeof(Real), cudaMemcpyDeviceToHost) );
  // iterate through to find the maximum inverse dt for this subgrid block
  for (int i=0; i<ngrid; i++) {
    max_dti = fmax(max_dti, host_dti_array[i]);
  }


  // add one to the counter
  block++;

}


  // free CPU memory
  free(host_dti_array);  
  free_buffers_3D(nx, ny, nz, nx_s, ny_s, nz_s, block1_tot, block2_tot, block3_tot, buffer);


  // free the GPU memory
  cudaFree(dev_conserved);
  cudaFree(dev_conserved_half);
  cudaFree(Q_Lx);
  cudaFree(Q_Rx);
  cudaFree(Q_Ly);
  cudaFree(Q_Ry);
  cudaFree(Q_Lz);
  cudaFree(Q_Rz);
  cudaFree(F_x);
  cudaFree(F_y);
  cudaFree(F_z);
  cudaFree(eta_x);
  cudaFree(eta_y);
  cudaFree(eta_z);
  cudaFree(etah_x);
  cudaFree(etah_y);
  cudaFree(etah_z);
  cudaFree(dev_dti_array);


  // return the maximum inverse timestep
  return max_dti;

}



__global__ void Update_Conserved_Variables_3D_half(Real *dev_conserved, Real *dev_conserved_half, Real *dev_F_x, Real *dev_F_y,  Real *dev_F_z,
                                              int nx, int ny, int nz, int n_ghost, Real dx, Real dy, Real dz, Real dt, Real gamma)
{

  int id, xid, yid, zid, n_cells;
  int imo, jmo, kmo;

  Real dtodx = dt/dx;
  Real dtody = dt/dy;
  Real dtodz = dt/dz;
  n_cells = nx*ny*nz;

  // get a global thread ID
  id = threadIdx.x + blockIdx.x * blockDim.x;
  zid = id / (nx*ny);
  yid = (id - zid*nx*ny) / nx;
  xid = id - zid*nx*ny - yid*nx;


  // threads corresponding to all cells except outer ring of ghost cells do the calculation
  if (xid > 0 && xid < nx-1 && yid > 0 && yid < ny-1 && zid > 0 && zid < nz-1)
  {
    // update the conserved variable array
    imo = xid-1 + yid*nx + zid*nx*ny;
    jmo = xid + (yid-1)*nx + zid*nx*ny;
    kmo = xid + yid*nx + (zid-1)*nx*ny;
    dev_conserved_half[            id] = dev_conserved[            id]
                                       + dtodx * (dev_F_x[            imo] - dev_F_x[            id])
                                       + dtody * (dev_F_y[            jmo] - dev_F_y[            id])
                                       + dtodz * (dev_F_z[            kmo] - dev_F_z[            id]);
    dev_conserved_half[  n_cells + id] = dev_conserved[  n_cells + id] 
                                       + dtodx * (dev_F_x[  n_cells + imo] - dev_F_x[  n_cells + id])
                                       + dtody * (dev_F_y[  n_cells + jmo] - dev_F_y[  n_cells + id])
                                       + dtodz * (dev_F_z[  n_cells + kmo] - dev_F_z[  n_cells + id]);
    dev_conserved_half[2*n_cells + id] = dev_conserved[2*n_cells + id] 
                                       + dtodx * (dev_F_x[2*n_cells + imo] - dev_F_x[2*n_cells + id])
                                       + dtody * (dev_F_y[2*n_cells + jmo] - dev_F_y[2*n_cells + id])
                                       + dtodz * (dev_F_z[2*n_cells + kmo] - dev_F_z[2*n_cells + id]);
    dev_conserved_half[3*n_cells + id] = dev_conserved[3*n_cells + id] 
                                       + dtodx * (dev_F_x[3*n_cells + imo] - dev_F_x[3*n_cells + id])
                                       + dtody * (dev_F_y[3*n_cells + jmo] - dev_F_y[3*n_cells + id])
                                       + dtodz * (dev_F_z[3*n_cells + kmo] - dev_F_z[3*n_cells + id]);
    dev_conserved_half[4*n_cells + id] = dev_conserved[4*n_cells + id] 
                                       + dtodx * (dev_F_x[4*n_cells + imo] - dev_F_x[4*n_cells + id])
                                       + dtody * (dev_F_y[4*n_cells + jmo] - dev_F_y[4*n_cells + id])
                                       + dtodz * (dev_F_z[4*n_cells + kmo] - dev_F_z[4*n_cells + id]);
    if (dev_conserved_half[id] < 0.0 || dev_conserved_half[id] != dev_conserved_half[id]) {
      printf("%3d %3d %3d Thread crashed in half step update.\n", xid, yid, zid);
    }    


  }

}


#endif //VL
#endif //CUDA
