
#include <hip/hip_runtime.h>
/*! \file CTU_2D_cuda.cu
 *  \brief Definitions of the cuda 2D CTU algorithm functions. */

#ifdef CUDA

#include<stdio.h>
#include<math.h>
#include<cuda.h>
#include"global.h"
#include"global_cuda.h"
#include"hydro_cuda.h"
#include"CTU_2D_cuda.h"
#include"pcm_cuda.h"
#include"plmp_ctu_cuda.h"
#include"plmc_ctu_cuda.h"
#include"ppmp_ctu_cuda.h"
#include"ppmc_ctu_cuda.h"
#include"exact_cuda.h"
#include"roe_cuda.h"
#include"h_correction_2D_cuda.h"
#include"cooling_cuda.h"
#include"subgrid_routines_2D.h"



__global__ void Evolve_Interface_States_2D(Real *dev_Q_Lx, Real *dev_Q_Rx, Real *dev_F1_x,
                                           Real *dev_Q_Ly, Real *dev_Q_Ry, Real *dev_F1_y,
                                           int nx, int ny, int n_ghost, Real dx, Real dy, Real dt);


Real CTU_Algorithm_2D_CUDA(Real *host_conserved, int nx, int ny, int n_ghost, Real dx, Real dy, Real dt)
{

  //Here, *host_conserved contains the entire
  //set of conserved variables on the grid
  //concatenated into a 1-d array

  #ifdef TIME
  // capture the start time
  cudaEvent_t start, stop;
  cudaEventCreate(&start);
  cudaEventCreate(&stop);
  float elapsedTime;
  #endif

  int n_fields = 5;
  #ifdef DE
  n_fields++;
  #endif


  // dimensions of subgrid blocks
  int nx_s; //number of cells in the subgrid block along x direction
  int ny_s; //number of cells in the subgrid block along y direction
  int nz_s = 1; //number of cells in the subgrid block along z direction

  // total number of blocks needed
  int block_tot;    //total number of subgrid blocks (unsplit == 1)
  int block1_tot;   //total number of subgrid blocks in x direction
  int block2_tot;   //total number of subgrid blocks in y direction
  int remainder1;   //modulus of number of cells after block subdivision in x direction
  int remainder2;   //modulus of number of cells after block subdivision in y direction 

  // counter for which block we're on
  int block = 0;

  // calculate the dimensions for each subgrid block
  sub_dimensions_2D(nx, ny, n_ghost, &nx_s, &ny_s, &block1_tot, &block2_tot, &remainder1, &remainder2, n_fields);
  block_tot = block1_tot*block2_tot;

  // number of cells in one subgrid block
  int BLOCK_VOL = nx_s*ny_s*nz_s;

  // define the dimensions for the 2D grid
  //int  ngrid = (n_cells + TPB - 1) / TPB;
  int  ngrid = (BLOCK_VOL + 2*TPB - 1) / (2*TPB);

  //number of blocks per 2-d grid  
  dim3 dim2dGrid(ngrid, 2, 1);

  //number of threads per 1-d block   
  dim3 dim1dBlock(TPB, 1, 1);

  // allocate buffer arrays to copy conserved variable slices into
  Real **buffer;
  allocate_buffers_2D(block1_tot, block2_tot, BLOCK_VOL, buffer, n_fields);
  // and set up pointers for the location to copy from and to
  Real *tmp1;
  Real *tmp2;

  // allocate an array on the CPU to hold max_dti returned from each thread block
  Real max_dti = 0;
  Real *host_dti_array;
  host_dti_array = (Real *) malloc(2*ngrid*sizeof(Real));

  // allocate GPU arrays
  // conserved variables
  Real *dev_conserved;
  // input states and associated interface fluxes (Q* and F* from Stone, 2008)
  Real *Q_Lx, *Q_Rx, *Q_Ly, *Q_Ry, *F_x, *F_y;
  // arrays to hold the eta values for the H correction
  Real *eta_x, *eta_y, *etah_x, *etah_y;
  // array of inverse timesteps for dt calculation
  Real *dev_dti_array;


  // allocate memory on the GPU
  CudaSafeCall( cudaMalloc((void**)&dev_conserved, n_fields*BLOCK_VOL*sizeof(Real)) );
  CudaSafeCall( cudaMalloc((void**)&Q_Lx, n_fields*BLOCK_VOL*sizeof(Real)) );
  CudaSafeCall( cudaMalloc((void**)&Q_Rx, n_fields*BLOCK_VOL*sizeof(Real)) );
  CudaSafeCall( cudaMalloc((void**)&Q_Ly, n_fields*BLOCK_VOL*sizeof(Real)) );
  CudaSafeCall( cudaMalloc((void**)&Q_Ry, n_fields*BLOCK_VOL*sizeof(Real)) );
  CudaSafeCall( cudaMalloc((void**)&F_x,  n_fields*BLOCK_VOL*sizeof(Real)) );
  CudaSafeCall( cudaMalloc((void**)&F_y,  n_fields*BLOCK_VOL*sizeof(Real)) );
  CudaSafeCall( cudaMalloc((void**)&eta_x,   BLOCK_VOL*sizeof(Real)) );
  CudaSafeCall( cudaMalloc((void**)&eta_y,   BLOCK_VOL*sizeof(Real)) );
  CudaSafeCall( cudaMalloc((void**)&etah_x,  BLOCK_VOL*sizeof(Real)) );
  CudaSafeCall( cudaMalloc((void**)&etah_y,  BLOCK_VOL*sizeof(Real)) );
  CudaSafeCall( cudaMalloc((void**)&dev_dti_array, 2*ngrid*sizeof(Real)) );


  // transfer first conserved variable slice into the first buffer
  host_copy_init_2D(nx, ny, nx_s, ny_s, n_ghost, block, block1_tot, remainder1, BLOCK_VOL, host_conserved, buffer, &tmp1, &tmp2, n_fields);
  

  while (block < block_tot) {

    // zero all the GPU arrays
    cudaMemset(dev_conserved, 0, n_fields*BLOCK_VOL*sizeof(Real));
    cudaMemset(Q_Lx,  0, n_fields*BLOCK_VOL*sizeof(Real));
    cudaMemset(Q_Rx,  0, n_fields*BLOCK_VOL*sizeof(Real));
    cudaMemset(Q_Ly,  0, n_fields*BLOCK_VOL*sizeof(Real));
    cudaMemset(Q_Ry,  0, n_fields*BLOCK_VOL*sizeof(Real));
    cudaMemset(F_x,   0, n_fields*BLOCK_VOL*sizeof(Real));
    cudaMemset(F_y,   0, n_fields*BLOCK_VOL*sizeof(Real));
    cudaMemset(eta_x,  0,  BLOCK_VOL*sizeof(Real));
    cudaMemset(eta_y,  0,  BLOCK_VOL*sizeof(Real));
    cudaMemset(etah_x, 0,  BLOCK_VOL*sizeof(Real));
    cudaMemset(etah_y, 0,  BLOCK_VOL*sizeof(Real));
    cudaMemset(dev_dti_array, 0, 2*ngrid*sizeof(Real));
    CudaCheckError();

    // copy the conserved variables onto the GPU
    CudaSafeCall( cudaMemcpy(dev_conserved, tmp1, n_fields*BLOCK_VOL*sizeof(Real), cudaMemcpyHostToDevice) );


    // Step 1: Do the reconstruction
    #ifdef PCM
    PCM_Reconstruction_2D<<<dim2dGrid,dim1dBlock>>>(dev_conserved, Q_Lx, Q_Rx, Q_Ly, Q_Ry, nx, ny, n_ghost, gama);
    #endif
    #ifdef PLMP
    PLMP_CTU<<<dim2dGrid,dim1dBlock>>>(dev_conserved, Q_Lx, Q_Rx, nx_s, ny_s, nz_s, n_ghost, dx, dt, gama, 0);
    PLMP_CTU<<<dim2dGrid,dim1dBlock>>>(dev_conserved, Q_Ly, Q_Ry, nx_s, ny_s, nz_s, n_ghost, dy, dt, gama, 1);
    #endif
    #ifdef PLMC
    PLMC_CTU<<<dim2dGrid,dim1dBlock>>>(dev_conserved, Q_Lx, Q_Rx, nx_s, ny_s, nz_s, n_ghost, dx, dt, gama, 0);
    PLMC_CTU<<<dim2dGrid,dim1dBlock>>>(dev_conserved, Q_Ly, Q_Ry, nx_s, ny_s, nz_s, n_ghost, dy, dt, gama, 1);
    #endif
    #ifdef PPMP
    PPMP_CTU<<<dim2dGrid,dim1dBlock>>>(dev_conserved, Q_Lx, Q_Rx, nx_s, ny_s, nz_s, n_ghost, dx, dt, gama, 0);
    PPMP_CTU<<<dim2dGrid,dim1dBlock>>>(dev_conserved, Q_Ly, Q_Ry, nx_s, ny_s, nz_s, n_ghost, dy, dt, gama, 1);
    #endif
    #ifdef PPMC
    PPMC_CTU<<<dim2dGrid,dim1dBlock>>>(dev_conserved, Q_Lx, Q_Rx, nx_s, ny_s, nz_s, n_ghost, dx, dt, gama, 0);
    PPMC_CTU<<<dim2dGrid,dim1dBlock>>>(dev_conserved, Q_Ly, Q_Ry, nx_s, ny_s, nz_s, n_ghost, dy, dt, gama, 1);
    #endif
    CudaCheckError();

    #ifdef H_CORRECTION
    #ifndef CTU
    // Step 3.5: Calculate eta values for H correction
    calc_eta_x_2D<<<dim2dGrid,dim1dBlock>>>(Q_Lx, Q_Rx, eta_x, nx_s, ny_s, n_ghost, gama);
    calc_eta_y_2D<<<dim2dGrid,dim1dBlock>>>(Q_Ly, Q_Ry, eta_y, nx_s, ny_s, n_ghost, gama);
    CudaCheckError();
    // and etah values for each interface
    calc_etah_x_2D<<<dim2dGrid,dim1dBlock>>>(eta_x, eta_y, etah_x, nx_s, ny_s, n_ghost);
    calc_etah_y_2D<<<dim2dGrid,dim1dBlock>>>(eta_x, eta_y, etah_y, nx_s, ny_s, n_ghost);
    CudaCheckError();
    #endif // NO CTU
    #endif // H_CORRECTION

    // Step 2: Calculate the fluxes
    #ifdef EXACT
    Calculate_Exact_Fluxes<<<dim2dGrid,dim1dBlock>>>(Q_Lx, Q_Rx, F_x, nx_s, ny_s, nz_s, n_ghost, gama, 0);
    Calculate_Exact_Fluxes<<<dim2dGrid,dim1dBlock>>>(Q_Ly, Q_Ry, F_y, nx_s, ny_s, nz_s, n_ghost, gama, 1);
    #endif
    #ifdef ROE
    Calculate_Roe_Fluxes<<<dim2dGrid,dim1dBlock>>>(Q_Lx, Q_Rx, F_x, nx_s, ny_s, nz_s, n_ghost, gama, etah_x, 0);
    Calculate_Roe_Fluxes<<<dim2dGrid,dim1dBlock>>>(Q_Ly, Q_Ry, F_y, nx_s, ny_s, nz_s, n_ghost, gama, etah_y, 1);
    #endif
    CudaCheckError();

#ifdef CTU

    // Step 3: Evolve the interface states
    Evolve_Interface_States_2D<<<dim2dGrid,dim1dBlock>>>(Q_Lx, Q_Rx, F_x, Q_Ly, Q_Ry, F_y, nx_s, ny_s, n_ghost, dx, dy, dt);
    CudaCheckError();


    #ifdef H_CORRECTION
    // Step 3.5: Calculate eta values for H correction
    calc_eta_x_2D<<<dim2dGrid,dim1dBlock>>>(Q_Lx, Q_Rx, eta_x, nx_s, ny_s, n_ghost, gama);
    calc_eta_y_2D<<<dim2dGrid,dim1dBlock>>>(Q_Ly, Q_Ry, eta_y, nx_s, ny_s, n_ghost, gama);
    CudaCheckError();
    // and etah values for each interface
    calc_etah_x_2D<<<dim2dGrid,dim1dBlock>>>(eta_x, eta_y, etah_x, nx_s, ny_s, n_ghost);
    calc_etah_y_2D<<<dim2dGrid,dim1dBlock>>>(eta_x, eta_y, etah_y, nx_s, ny_s, n_ghost);
    CudaCheckError();
    #endif


    // Step 4: Calculate the fluxes again
    #ifdef EXACT
    Calculate_Exact_Fluxes<<<dim2dGrid,dim1dBlock>>>(Q_Lx, Q_Rx, F_x, nx_s, ny_s, nz_s, n_ghost, gama, 0);
    Calculate_Exact_Fluxes<<<dim2dGrid,dim1dBlock>>>(Q_Ly, Q_Ry, F_y, nx_s, ny_s, nz_s, n_ghost, gama, 1);
    #endif
    #ifdef ROE
    Calculate_Roe_Fluxes<<<dim2dGrid,dim1dBlock>>>(Q_Lx, Q_Rx, F_x, nx_s, ny_s, nz_s, n_ghost, gama, etah_x, 0);
    Calculate_Roe_Fluxes<<<dim2dGrid,dim1dBlock>>>(Q_Ly, Q_Ry, F_y, nx_s, ny_s, nz_s, n_ghost, gama, etah_y, 1);
    #endif
    CudaCheckError();

#endif //CTU


    // Step 5: Update the conserved variable array
    Update_Conserved_Variables_2D<<<dim2dGrid,dim1dBlock>>>(dev_conserved, F_x, F_y, nx_s, ny_s, n_ghost, dx, dy, dt, gama);
    CudaCheckError();

    // Synchronize the total and internal energy
    #ifdef DE
    Sync_Energies_2D<<<dim2dGrid,dim1dBlock>>>(dev_conserved, nx_s, ny_s, n_ghost, gama);
    CudaCheckError();    
    #endif

    // Apply cooling
    #ifdef COOLING_GPU
    cooling_kernel<<<dim2dGrid,dim1dBlock>>>(dev_conserved, nx_s, ny_s, nz_s, n_ghost, dt, gama);
    CudaCheckError();    
    #endif

    // Step 6: Calculate the next timestep
    Calc_dt_2D<<<dim2dGrid,dim1dBlock>>>(dev_conserved, nx_s, ny_s, n_ghost, dx, dy, dev_dti_array, gama);
    CudaCheckError();    


    // copy the conserved variable array back to the CPU
    CudaSafeCall( cudaMemcpy(tmp2, dev_conserved, n_fields*BLOCK_VOL*sizeof(Real), cudaMemcpyDeviceToHost) );

    // copy the next conserved variable blocks into appropriate buffers
    host_copy_next_2D(nx, ny, nx_s, ny_s, n_ghost, block, block1_tot, block2_tot, remainder1, remainder2, BLOCK_VOL, host_conserved, buffer, &tmp1, n_fields);

    // copy the updated conserved variable array back into the host_conserved array on the CPU
    host_return_values_2D(nx, ny, nx_s, ny_s, n_ghost, block, block1_tot, block2_tot, remainder1, remainder2, BLOCK_VOL, host_conserved, buffer, n_fields);


    // copy the dti array onto the CPU
    CudaSafeCall( cudaMemcpy(host_dti_array, dev_dti_array, 2*ngrid*sizeof(Real), cudaMemcpyDeviceToHost) );
    // iterate through to find the maximum inverse dt for this subgrid block
    for (int i=0; i<2*ngrid; i++) {
      max_dti = fmax(max_dti, host_dti_array[i]);
    }


    // add one to the counter
    block++;

  }


  // free the CPU memory
  free(host_dti_array);
  free_buffers_2D(nx, ny, nx_s, ny_s, block1_tot, block2_tot, buffer);

  // free the GPU memory
  cudaFree(dev_conserved);
  cudaFree(Q_Lx);
  cudaFree(Q_Rx);
  cudaFree(Q_Ly);
  cudaFree(Q_Ry);
  cudaFree(F_x);
  cudaFree(F_y);
  cudaFree(eta_x);
  cudaFree(eta_y);
  cudaFree(etah_x);
  cudaFree(etah_y);
  cudaFree(dev_dti_array);


  // return the maximum inverse timestep
  return max_dti;

}





__global__ void Evolve_Interface_States_2D(Real *dev_Q_Lx, Real *dev_Q_Rx, Real *dev_F_x, 
                                           Real *dev_Q_Ly, Real *dev_Q_Ry, Real *dev_F_y,
                                           int nx, int ny, int n_ghost, Real dx, Real dy, Real dt)
{
  Real dtodx = dt/dx;
  Real dtody = dt/dy;
  int n_cells = nx*ny;

  // get a thread ID
  int blockId = blockIdx.x + blockIdx.y*gridDim.x;
  int tid = threadIdx.x + blockId * blockDim.x;
  int yid = tid / nx;
  int xid = tid - yid*nx;
  int id = xid + yid*nx;


  // set the new x interface states
  if (xid > n_ghost-2 && xid < nx-n_ghost && yid > n_ghost-2 && yid < ny-n_ghost+1)
  {
    // left
    int ipo = xid+1 + yid*nx;
    int jmo = xid + (yid-1)*nx;
    int ipojmo = xid+1 + (yid-1)*nx;
    dev_Q_Lx[            id] += 0.5*dtody*(dev_F_y[            jmo] - dev_F_y[            id]);
    dev_Q_Lx[  n_cells + id] += 0.5*dtody*(dev_F_y[  n_cells + jmo] - dev_F_y[  n_cells + id]);
    dev_Q_Lx[2*n_cells + id] += 0.5*dtody*(dev_F_y[2*n_cells + jmo] - dev_F_y[2*n_cells + id]);
    dev_Q_Lx[3*n_cells + id] += 0.5*dtody*(dev_F_y[3*n_cells + jmo] - dev_F_y[3*n_cells + id]);
    dev_Q_Lx[4*n_cells + id] += 0.5*dtody*(dev_F_y[4*n_cells + jmo] - dev_F_y[4*n_cells + id]);
    // right
    dev_Q_Rx[            id] += 0.5*dtody*(dev_F_y[            ipojmo] - dev_F_y[            ipo]);
    dev_Q_Rx[  n_cells + id] += 0.5*dtody*(dev_F_y[  n_cells + ipojmo] - dev_F_y[  n_cells + ipo]);
    dev_Q_Rx[2*n_cells + id] += 0.5*dtody*(dev_F_y[2*n_cells + ipojmo] - dev_F_y[2*n_cells + ipo]);
    dev_Q_Rx[3*n_cells + id] += 0.5*dtody*(dev_F_y[3*n_cells + ipojmo] - dev_F_y[3*n_cells + ipo]);
    dev_Q_Rx[4*n_cells + id] += 0.5*dtody*(dev_F_y[4*n_cells + ipojmo] - dev_F_y[4*n_cells + ipo]);
  }
  // set the new y interface states
  if (yid > n_ghost-2 && yid < ny-n_ghost && xid > n_ghost-2 && xid < nx-n_ghost+1)
  {
    // left
    int jpo = xid + (yid+1)*nx;
    int imo = xid-1 + yid*nx;
    int jpoimo = xid-1 + (yid+1)*nx;
    dev_Q_Ly[            id] += 0.5*dtodx*(dev_F_x[            imo] - dev_F_x[            id]); 
    dev_Q_Ly[  n_cells + id] += 0.5*dtodx*(dev_F_x[  n_cells + imo] - dev_F_x[  n_cells + id]); 
    dev_Q_Ly[2*n_cells + id] += 0.5*dtodx*(dev_F_x[2*n_cells + imo] - dev_F_x[2*n_cells + id]); 
    dev_Q_Ly[3*n_cells + id] += 0.5*dtodx*(dev_F_x[3*n_cells + imo] - dev_F_x[3*n_cells + id]); 
    dev_Q_Ly[4*n_cells + id] += 0.5*dtodx*(dev_F_x[4*n_cells + imo] - dev_F_x[4*n_cells + id]); 
    // right
    dev_Q_Ry[            id] += 0.5*dtodx*(dev_F_x[            jpoimo] - dev_F_x[            jpo]); 
    dev_Q_Ry[  n_cells + id] += 0.5*dtodx*(dev_F_x[  n_cells + jpoimo] - dev_F_x[  n_cells + jpo]); 
    dev_Q_Ry[2*n_cells + id] += 0.5*dtodx*(dev_F_x[2*n_cells + jpoimo] - dev_F_x[2*n_cells + jpo]); 
    dev_Q_Ry[3*n_cells + id] += 0.5*dtodx*(dev_F_x[3*n_cells + jpoimo] - dev_F_x[3*n_cells + jpo]); 
    dev_Q_Ry[4*n_cells + id] += 0.5*dtodx*(dev_F_x[4*n_cells + jpoimo] - dev_F_x[4*n_cells + jpo]); 
  }

}


#endif //CUDA

